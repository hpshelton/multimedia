//http://www.embl.de/~gpau/index.html

#include <cutil_inline.h>
#include "kernels.cu"

extern "C" void CUquantize(float* x, int Qlevel, int maxval, int len);
extern "C" void CUzeroOut(int* x, float threshold, int len);
extern "C" void CUtranspose(float* d_odata, float* d_idata, int col, int row);
extern "C" void CUsetToVal(unsigned char* x, int len, int val);
extern "C" void CUedgeDetect(unsigned char* input, unsigned char* output, int row, int col);
extern "C" void CUblur(unsigned char* output, unsigned char* input, int row, int col);
extern "C" void CUbrighten(unsigned char* output, unsigned char* input, int row, int col, float factor);
extern "C" void CUcontrast(unsigned char* output, unsigned char* input, int row, int col, float factor, float lum);
extern "C" void CUgreyscale(unsigned char* output, unsigned char* input, int row, int col);
extern "C" void CUsaturate(unsigned char* output, unsigned char* input, int row, int col, float factor);
extern "C" void CUfwt97_2D_rgba(int* outputInt, unsigned char* input, int row, int col);
extern "C" void CUiwt97_2D_rgba(unsigned char* output, int* input, int row, int col);

void CUquantize(float* x, int Qlevel, int maxval, int len)
{
	int threadsPerBlock = 512;
	int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
	quantize<<<blocksPerGrid, threadsPerBlock>>>(x, Qlevel, maxval, len);
}

void CUzeroOut(int* x, float threshold, int len)
{
	int threadsPerBlock = 512;
	int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
	zeroOut<<<blocksPerGrid, threadsPerBlock>>>(x, threshold, len);
}

void CUtranspose(float* d_odata, float* d_idata, int col, int row)
{
	dim3 grid(col / BLOCK_DIM, row / BLOCK_DIM, 1);
	dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
	transpose<<< grid, threads >>>(d_odata, d_idata, col, row);
}

void CUsetToVal(unsigned char* x, int len, int val)
{
	int threadsPerBlock = len;
	int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
	setToVal<<<blocksPerGrid, threadsPerBlock>>>(x, len, val);
}

void CUedgeDetect(unsigned char* output, unsigned char* input, int row, int col)
{
	dim3 dimGrid(row/4+1, col/4+1);
	dim3 dimThreadBlock(16,16);

	float coeff[9]= {-1, -1, -1, \
					 -1,  8, -1, \
					 -1, -1, -1};
	float* CUcoeff;
	cutilSafeCall(hipMalloc((void**)&CUcoeff, sizeof(float)*9));
	cutilSafeCall(hipMemcpy(CUcoeff, coeff, sizeof(float)*9, hipMemcpyHostToDevice));

	conv3x3<<<dimGrid, dimThreadBlock>>>(input, output, row, col, CUcoeff);
	cutilSafeCall(hipFree(CUcoeff));
}

void CUblur(unsigned char* output, unsigned char* input, int row, int col)
{
	dim3 dimGrid(row/4+1, col/4+1);
	dim3 dimThreadBlock(16,16);

	float coeff[9]= { 0.0625, 0.125, 0.0625, \
					  0.125,  0.25,  0.125,  \
					  0.0625, 0.125, 0.0625 };

	float* CUcoeff;
	cutilSafeCall(hipMalloc((void**)&CUcoeff, sizeof(float)*9));
	cutilSafeCall(hipMemcpy(CUcoeff, coeff, sizeof(float)*9, hipMemcpyHostToDevice));

	conv3x3<<<dimGrid, dimThreadBlock>>>(input, output, row, col, CUcoeff);
	cutilSafeCall(hipFree(CUcoeff));
}

void CUbrighten(unsigned char* output, unsigned char* input, int row, int col, float factor)
{
	dim3 dimGrid(row/4+1, col/4+1);
	dim3 dimThreadBlock(16,16);
	brighten<<<dimGrid, dimThreadBlock>>>(input, output, row, col, factor);
}

void CUcontrast(unsigned char* output, unsigned char* input, int row, int col, float factor, float lum)
{
	dim3 dimGrid(row/4+1, col/4+1);
	dim3 dimThreadBlock(16,16);
	contrast<<<dimGrid, dimThreadBlock>>>(input, output, row, col, factor, lum);
}

void CUgreyscale(unsigned char* output, unsigned char* input, int row, int col)
{
	dim3 dimGrid(row/16+1, col/16+1);
	dim3 dimThreadBlock(16,16);
	greyscale<<<dimGrid, dimThreadBlock>>>(input, output, row, col);
}

void CUsaturate(unsigned char* output, unsigned char* input, int row, int col, float factor)
{
	dim3 dimGrid(row/16+1, col/16+1);
	dim3 dimThreadBlock(16,16);
	saturate<<<dimGrid, dimThreadBlock>>>(input, output, row, col, factor);
}

void fwt2D_row(float* output, float* tempbank, int n, int len, int dim, dim3 numBlocks, dim3 threadsPerBlock, int row, int col)
{
	predict<<<numBlocks, threadsPerBlock>>>(output, n, len, dim, -1.586134342f, col, row);
	update <<<numBlocks, threadsPerBlock>>>(output, n, len, dim, -0.05298011854f, col, row);
	predict<<<numBlocks, threadsPerBlock>>>(output, n, len, dim,  0.8829110762f, col, row);
	update <<<numBlocks, threadsPerBlock>>>(output, n, len, dim,  0.4435068522, col, row);
	scale  <<<numBlocks, threadsPerBlock>>>(output, n, len, dim,  0.869864452f, col, row);
	pack   <<<numBlocks, threadsPerBlock>>>(output, tempbank, n, len, dim);
	readOut<<<numBlocks, threadsPerBlock>>>(output, tempbank, n, len, dim);
}

void iwt2D_row(float* input, float* tempbank, int n, int len, int dim, dim3 numBlocks, dim3 threadsPerBlock, int row, int col)
{
	unpack <<<numBlocks, threadsPerBlock>>>(input, tempbank,n,len, dim);
	readOut<<<numBlocks, threadsPerBlock>>>(input, tempbank,n,len, dim);
	scale  <<<numBlocks, threadsPerBlock>>>(input, n, len, dim,  1.149604398f, col, row);
	update <<<numBlocks, threadsPerBlock>>>(input, n, len, dim, -0.4435068522f, col, row);
	predict<<<numBlocks, threadsPerBlock>>>(input, n, len, dim, -0.8829110762f, col, row);
	update <<<numBlocks, threadsPerBlock>>>(input, n, len, dim,  0.05298011854f, col, row);
	predict<<<numBlocks, threadsPerBlock>>>(input, n, len, dim,  1.586134342f, col, row);
}

void CUfwt97_2D_rgba(int* outputInt, unsigned char* input, int row, int col)
{
/*	if(row%2)
		row++;
	if(col%2)
		col++;
*/
	dim3 numBlocks(row/8+1,col/8+1,1);
	dim3 threadsPerBlock(8,8,4);
	int dim = row;

	float* tempbank;
	float* outputT;
	float* output;
	cutilSafeCall(hipMalloc((void**)&tempbank, sizeof(float) * row*col*4));
	cutilSafeCall(hipMalloc((void**)&outputT,  sizeof(float) * row*col*4));
	cutilSafeCall(hipMalloc((void**)&output,   sizeof(float) * row*col*4));

	int threads = 512;
	int blocks = (row*col*4 + threads - 1) / threads;
	shuffle<<<blocks,threads>>>(outputT, input, col, row);


//	setToVal<<<blocks, threads>>>(output, row*col*4, 255);
//	setToVal<<<blocks, threads>>>(outputT, row*col*4, 100);
//	hipError_t err;
//	err = hipMemcpy(output, outputT, sizeof(float)*row*col*4, hipMemcpyDeviceToDevice);
//	printf("%d\n",err);
//	fflush(stdout);
//	cutilSafeCall(hipMemcpy(output, outputR, sizeof(float)*row*col*4, hipMemcpyDeviceToDevice));

	// execute the kernel
	fwt2D_row(outputT, tempbank, row*col*4, col, dim, numBlocks, threadsPerBlock, col,row);

	CUtranspose(&output[0],         &outputT[0], col,row);
	CUtranspose(&output[row*col*1], &outputT[row*col*1], col,row);
	CUtranspose(&output[row*col*2], &outputT[row*col*2], col,row);
	CUtranspose(&output[row*col*3], &outputT[row*col*3], col,row);

	fwt2D_row(output, tempbank, row*col*4, row, dim, numBlocks, threadsPerBlock, col,row);
#ifdef TWODFWT
	CUtranspose(&outputT[0],         &output[0], row,col);
	CUtranspose(&outputT[row*col*1], &output[row*col*1], row,col);
	CUtranspose(&outputT[row*col*2], &output[row*col*2], row,col);
	CUtranspose(&outputT[row*col*3], &output[row*col*3], row,col);

	fwt2D_row(outputT, tempbank, row*col*4, col, dim, numBlocks, threadsPerBlock, col,row);

	CUtranspose(&output[0],         &outputT[0], col,row);
	CUtranspose(&output[row*col*1], &outputT[row*col*1], col,row);
	CUtranspose(&output[row*col*2], &outputT[row*col*2], col,row);
	CUtranspose(&output[row*col*3], &outputT[row*col*3], col,row);

	fwt2D_row(output, tempbank, row*col*4, row, dim, numBlocks, threadsPerBlock, col,row);
#endif
	roundArray<<<blocks,threads>>>(outputInt, output, col, row);

	cutilSafeCall(hipFree(tempbank));
	cutilSafeCall(hipFree(outputT));
	cutilSafeCall(hipFree(output));
}

void CUiwt97_2D_rgba(unsigned char* output, int* inputInt, int row, int col)
{
/*	if(row%2)
		row++;
	if(col%2)
		col++;
*/
	dim3 numBlocks(row/8+1,col/8+1,1);
	dim3 threadsPerBlock(8,8,4);
	int dim = row;

	float* tempbank;
	float* input;
	float* inputT;
	cutilSafeCall(hipMalloc((void**)&tempbank, sizeof(float) * row*col*4));
	cutilSafeCall(hipMalloc((void**)&input, sizeof(float)*row*col*4));
	cutilSafeCall(hipMalloc((void**)&inputT, sizeof(float)*row*col*4));

	int threads = 512;
	int blocks = (row*col*4 + threads - 1) / threads;
	intToFloat<<<blocks,threads>>>(input, inputInt, row,col);

	// execute the kernel
	iwt2D_row(input, tempbank, row*col*4,row,dim,numBlocks, threadsPerBlock, col,row);

	CUtranspose(&inputT[0]        , &input[0], row,col);
	CUtranspose(&inputT[row*col*1], &input[row*col*1], row,col);
	CUtranspose(&inputT[row*col*2], &input[row*col*2], row,col);
	CUtranspose(&inputT[row*col*3], &input[row*col*3], row,col);

	iwt2D_row(inputT, tempbank,row*col*4,col,dim,numBlocks, threadsPerBlock, col,row);
#ifdef TWODFWT
	CUtranspose(&input[0]        , &inputT[0], col,row);
	CUtranspose(&input[row*col*1], &inputT[row*col*1], col,row);
	CUtranspose(&input[row*col*2], &inputT[row*col*2], col,row);
	CUtranspose(&input[row*col*3], &inputT[row*col*3], col,row);

	iwt2D_row(input, tempbank, row*col*4,row,dim,numBlocks, threadsPerBlock, col,row);

	CUtranspose(&inputT[0]        , &input[0], row,col);
	CUtranspose(&inputT[row*col*1], &input[row*col*1], row,col);
	CUtranspose(&inputT[row*col*2], &input[row*col*2], row,col);
	CUtranspose(&inputT[row*col*3], &input[row*col*3], row,col);

	iwt2D_row(inputT, tempbank,row*col*4,col,dim,numBlocks, threadsPerBlock, col,row);
#endif
	UNshuffle<<<blocks,threads>>>(output, inputT, col, row);

	cutilSafeCall(hipFree(input));
	cutilSafeCall(hipFree(inputT));
	cutilSafeCall(hipFree(tempbank));
}
