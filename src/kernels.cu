#include "hip/hip_runtime.h"
#include "mvec.h"
#include "defines.h"

__global__ void setToVal(unsigned char* x, int len, int val)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if(index < len)
		x[index] = val;
}

__global__ void setToVal(float* x, int len, int val)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if(index < len)
		x[index] = val;
}

__global__ void conv3x3(unsigned char* input, unsigned char* output, int row, int col, float* kernel)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int index = (xIndex + yIndex * row*4);

	if(index < row*col*4){
		int i, j;
		float convSum=0;
		for(i=-1; i < 2; i++){
			for(j=-1; j < 2; j++){
				if(-1 < (index+4*j)+(4*col*i) && (index+4*j)+(4*col*i) < row*col*4){
				convSum += kernel[3*(i+1) + (j+1)]*input[(index+4*j)+(4*col*i)];
				}
			}
		}
		output[index] = CLAMP(convSum);
	}
}

__global__ void quantize(float* x, int Qlevel, float maxval, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<n)
		x[i] = (int)( (x[i]/maxval) *Qlevel ) * (maxval / (float)Qlevel);
}

__global__ void shuffle(float* output, unsigned char* input, int width, int height)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<width*height*4)
		output[i/4 + (i%4)*width*height] = input[i];
}
__global__ void UNshuffle(unsigned char* output, float* input, int width, int height)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<width*height*4)
		output[i] = CLAMP(input[i/4 + (i%4)*width*height]);
}

#define ABS(a) (a<0?-a:a)


__global__ void zeroOut(int* x, float threshold, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<n){
		if(ABS(x[i]) < threshold)
			x[i]=0;
	}
}


__global__ void brighten(unsigned char* input, unsigned char* output, int row, int col, float factor)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int index = (xIndex + yIndex * row*4);

	if(index < row*col*4){
		output[index]=CLAMP(factor*input[index]);
	}
}

__global__ void greyscale(unsigned char* input, unsigned char* output, int row, int col)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int index = (4*xIndex + yIndex * row*4);

	if(index < row*col*4){
		int lum = 0.11*input[index] + 0.59*input[index+1] + 0.3*input[index+2];

		output[index]=lum;
		output[index+1]=lum;
		output[index+2]=lum;
		output[index+3]=0;
	}
}

__global__ void saturate(unsigned char* input, unsigned char* output, int row, int col, float factor)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int index = (4*xIndex + yIndex * row*4);

	if(index < row*col*4){
		int lum = 0.11*input[index] + 0.59*input[index+1] + 0.3*input[index+2];

		output[index]=CLAMP( (1-factor)*lum + factor*input[index] );
		output[index+1]=CLAMP( (1-factor)*lum + factor*input[index+1] );
		output[index+2]=CLAMP( (1-factor)*lum + factor*input[index+2] );
		output[index+3]=0;
	}
}

__global__ void contrast(unsigned char* input, unsigned char* output, int row, int col, float factor, float lum)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int index = (4*xIndex + yIndex * row*4);

	if(index < row*col*4){
		output[index]=CLAMP( (1-factor)*lum + factor*input[index] );
		output[index+1]=CLAMP( (1-factor)*lum + factor*input[index+1] );
		output[index+2]=CLAMP( (1-factor)*lum + factor*input[index+2] );
		output[index+3]=0;
	}
}

__global__ void predict(float* input, int n, int len, int dim, float a, int width, int height)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int color = threadIdx.z*(n/4);
	int i = (xIndex + yIndex * dim);

	if (xIndex < width && yIndex < height && i+color < n)
	{
		if(i%2==1){
			if((i%len)!=(len-1)){
				input[color+i]+=a*(input[color+i-1]+input[color+i+1]);
			}
			else{
				input[color+i]+=2*a*input[color+i-1];
			}
		}
	}
}

__global__ void update(float* input, int n, int len, int dim, float a, int width, int height)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int i = (xIndex + yIndex * dim);
	int color = threadIdx.z*(n/4);

	if (xIndex < width && yIndex < height && i+color < n)
	{
		if(i%2==0){
			if((i%len)!=0 && (i%len)!=len-1){
				input[color+i]+=a*(input[color+i-1]+input[color+i+1]);
			}
			else if(i%len==0){
				input[color+i]+=2*a*input[color+i+1];
			}
			else{
				input[color+i]+=2*a*input[color+i-1];
			}
		}
	}
}

__global__ void scale(float* input, int n, int len, int dim, float a, int width, int height)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if (xIndex < width && yIndex < height && (xIndex + yIndex * dim) + threadIdx.z*(n/4) < n)
	{
	int i = (xIndex + yIndex * dim) + threadIdx.z*(n/4);
		if (i%2)
			input[i] = input[i]*a;
		else
			input[i] = input[i]/a;
	}
}

__global__ void pack(float* input, float* tempbank, int n, int len, int dim)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int i = (xIndex + yIndex * dim);
	int color = threadIdx.z*(n/4);

	if ((i+color)<n)
	{
		int rowNum = i / len;
		int rowIndex = i%len;
		if (i%2==0)
			tempbank[color + len*rowNum + rowIndex/2] = input[color+i];
		else
			tempbank[color + len*rowNum + rowIndex/2 + len/2] = input[color+i];
	}
}

__global__ void unpack(float* input, float* tempbank, int n, int len, int dim)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int i = (xIndex + yIndex * dim);
	int color = threadIdx.z*(n/4);

	if ((i+color)<n)
	{
		int rowNum = i / len;
		int rowIndex = i%len;
		if(i%2==0)
			tempbank[color+i] = input[color + len*rowNum + rowIndex/2];
		else
			tempbank[color+i] = input[color + len*rowNum + rowIndex/2 + len/2];
	}
}

__global__ void readOut(float* input, float* tempbank, int n, int len, int dim)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	int i = (xIndex + yIndex * dim)+threadIdx.z*(n/4);;

	if ((i)<n)
		input[i]=tempbank[i];
}

__global__ void roundArray(int* output, float* input, int width, int height)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<width*height*4)
		output[i] = (int)(input[i]+0.5);
}

__global__ void intToFloat(float* output, int* input, int width, int height)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<width*height*4)
		output[i] = input[i];
}

#define BLOCK_DIM 16

__global__ void transpose(float *odata, float *idata, int width, int height)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T> struct SharedMemory
{
	__device__ inline operator	   T*()
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}

	__device__ inline operator const T*() const
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
};

template <class T> __global__ void reduce3(T *g_idata, T *g_odata, unsigned int n)
{
	T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	if(i < n)
		sdata[tid] = g_idata[i];
	else{
		sdata[tid].x = 0;
		sdata[tid].y = 0;
		sdata[tid].diff = INT_MAX;
	}

	if (i + blockDim.x < n)
		sdata[tid] += g_idata[i+blockDim.x];

	__syncthreads();

	// do reduction in shared mem
	for(unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void findAllVals(mvec* in, int numYblocks, int numXblocks, short int* prevImg, unsigned char* currImg, int height, int width)
{
	int index = blockDim.x*blockDim.y*(blockIdx.x + blockIdx.y * numXblocks) + threadIdx.x + threadIdx.y*blockDim.x;
	int threadsPerBlock = blockDim.x * blockDim.y;

	if(index < numYblocks*numXblocks * threadsPerBlock)
	{
//		int vecBlockID = blockIdx.x + blockIdx.y * numXblocks;
		int shiftedXIndex, shiftedYIndex, xIndex, yIndex;
		in[index].x = (threadIdx.x-8)*4;
		in[index].y = threadIdx.y-8;
		in[index].diff=0;
		for(int i=0; i < 32; i++){
			for(int j=0; j < 8; j++){
				shiftedXIndex = blockIdx.x * blockDim.x + i + (threadIdx.x-8)*4;
				shiftedYIndex = blockIdx.y * blockDim.y + j + (threadIdx.y-8);
				xIndex = blockIdx.x * blockDim.x + i;
				yIndex = blockIdx.y * blockDim.y + j;

				if(shiftedXIndex < 0 || shiftedXIndex >= width*4 || shiftedYIndex < 0 || shiftedYIndex >= height)
					in[index].diff +=     currImg[xIndex + yIndex * width*4];
				else
					in[index].diff += abs(currImg[xIndex + yIndex * width*4] - prevImg[shiftedXIndex + shiftedYIndex * width*4]);
			}
		}
	}
}
